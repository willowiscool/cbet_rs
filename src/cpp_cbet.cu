#include "hip/hip_runtime.h"
#include "cbet_rs/include/cpp_cbet.cuh"
#include "cbet_rs/src/cbet.rs.h"
#include <stdio.h>
#include <algorithm>
#include <cmath>
#include <cstddef>

#define CEIL_DIV(a, b) ((a+b-1)/b)

// TODO: MOVE TO USE RUST VALUES!!!
// [brc]n = beam/ray/crossing number
// n[brc] = number of beams/rays/crossings

const double CONVERGE = 1e-7;
const double MAX_INCR = 0.2;
const double CBETCONVERGENCE = 0.9990;

// headers that can't go in .h file bc compiler reasons
__global__ void get_cbet_gain(CbetCrossing* cbet_crossings, CbetCrosses* cbet_crosses, double* w_mult_values, size_t nb, size_t nr, size_t nc);
__global__ void update_intensities(CbetCrossing* cbet_crossings, double* w_mult_values, size_t nb, size_t nr, size_t nc, double* conv_max, double curr_max);

void cpp_cbet(CbetCrossing* cbet_crossings, CbetCrosses* cbet_crosses, size_t nb, size_t nr, size_t nc) {
	printf("Running cbet in cpp\n");
	// create w_mult_values struct:
	//double* w_mult_values = new double[nb*nr*nc];

	CbetCrossing* cuda_cbet_crossings;
	CbetCrosses* cuda_cbet_crosses;
	double* cuda_w_mult_values;
	double* cuda_conv_max;
	double conv_max = 0.0;
	hipMalloc(&cuda_cbet_crossings, nb*nr*nc * sizeof(CbetCrossing));
	hipMalloc(&cuda_cbet_crosses, nb*nr*nc*nb * sizeof(CbetCrosses));
	hipMalloc(&cuda_w_mult_values, nb*nr*nc * sizeof(double));
	hipMalloc(&cuda_conv_max, sizeof(double));
	hipMemcpy(cuda_cbet_crossings, cbet_crossings, nb*nr*nc * sizeof(CbetCrossing), hipMemcpyHostToDevice);
	hipMemcpy(cuda_cbet_crosses, cbet_crosses, nb*nr*nc*nb * sizeof(CbetCrosses), hipMemcpyHostToDevice);
	hipMemcpy(cuda_conv_max, &conv_max, sizeof(double), hipMemcpyHostToDevice);

	// let's say each thread processes a single ray
	// there are nb*nr rays
	dim3 threads_per_block(32, 32);
	int blocks = CEIL_DIV(nb*nr, 1024);

	double currmax = MAX_INCR;
	for (size_t i = 1; i <= 500; i++) {
		conv_max = 0.0;
		hipMemcpy(cuda_conv_max, &conv_max, sizeof(double), hipMemcpyHostToDevice);

		get_cbet_gain<<<blocks, threads_per_block>>>(cuda_cbet_crossings, cuda_cbet_crosses, cuda_w_mult_values, nb, nr, nc);
		update_intensities<<<blocks, threads_per_block>>>(cuda_cbet_crossings, cuda_w_mult_values, nb, nr, nc, cuda_conv_max, currmax);

		hipMemcpy(&conv_max, cuda_conv_max, sizeof(double), hipMemcpyDeviceToHost);
		if (conv_max <= CONVERGE) {
			break;
		}

		double currmaxa = MAX_INCR*pow(CBETCONVERGENCE, i);
		double currmaxb = CBETCONVERGENCE*conv_max;
		currmax = std::min(currmaxa, currmaxb);
	}

	hipMemcpy(cbet_crossings, cuda_cbet_crossings, nb*nr*nc * sizeof(CbetCrossing), hipMemcpyDeviceToHost);
}

__global__ void get_cbet_gain(CbetCrossing* cbet_crossings, CbetCrosses* cbet_crosses, double* w_mult_values, size_t nb, size_t nr, size_t nc) {
	// ray index = beam number * nr + ray number
	size_t ray_index = (((blockIdx.x*1024)+threadIdx.y)*32)+threadIdx.x;
	if (ray_index > nb*nr) return;
	for (size_t cn = 0; cn < nc; cn++) {
		// should take address rather than value?
		CbetCrossing crossing = cbet_crossings[(ray_index*nc)+cn];
		if (crossing.intensity == 0.0) break;
		double cbet_sum = 0.0;
		for (size_t crossn = 0; crossn < nb; crossn++) {
			// xs = crosses
			CbetCrosses xs = cbet_crosses[(((ray_index*nc)+cn)*nb)+crossn];
			if (xs.coupling_mult == 0.0) break;
			double other_intensity1 = cbet_crossings[(((xs.b_num*nr)+xs.r_num)*nc)+xs.c_num].intensity;
			double other_intensity2 = cbet_crossings[(((xs.b_num*nr)+xs.r_num)*nc)+xs.c_num_next].intensity;
			double avg_intensity = (other_intensity1+other_intensity2)/2.0;
			cbet_sum += avg_intensity*xs.coupling_mult;
		}
		w_mult_values[(ray_index*nc)+cn] = exp(-1.0*cbet_sum) * crossing.absorption_coeff;
	}
}

// changes conv max to a pointer!!! writes into it!!!
__global__ void update_intensities(CbetCrossing* cbet_crossings, double* w_mult_values, size_t nb, size_t nr, size_t nc, double* conv_max, double curr_max) {
	size_t ray_index = (((blockIdx.x*1024)+threadIdx.y)*32)+threadIdx.x;
	if (ray_index > nb*nr) return;
	double i0 = cbet_crossings[(ray_index*nc)+0].intensity;
	double mult_acc = 1.0;
	for (size_t cn = 0; cn < nc; cn++) {
		CbetCrossing crossing = cbet_crossings[(ray_index*nc)+cn];
		if (crossing.intensity == 0.0) break;
		double i_curr = i0*mult_acc;
		double fractional_change = abs(i_curr-crossing.intensity)/crossing.intensity;
		*conv_max = max(fractional_change, *conv_max);
		if (fractional_change > curr_max) {
			int sign = i_curr - crossing.intensity > 0.0 ? 1.0 : -1.0;
			double correction = 1.0 + curr_max*sign;
			i_curr = crossing.intensity*correction;
		}
		mult_acc *= w_mult_values[(ray_index*nc)+cn];
		crossing.intensity = i_curr;
		cbet_crossings[(ray_index*nc)+cn] = crossing;
	}
}
